
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template<typename IT, typename OT, size_t N>
__global__ void cast(IT* input, OT* output) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N){
        output[tid] = static_cast<OT>(input[tid]);
    }
}